#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

#define SOFTENING 1e-9f
#define BLOCK_SIZE 128
#define BLOCK_TILE 32
#define BLOCK_NUM 32

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

extern __shared__ float sp[];
__global__ void bodyForce(Body *p, int *state, float dt, int n) {
    int i = (threadIdx.x + blockDim.x * blockIdx.x) % n;

    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
    float sx0 = p[i].x; float sy0 = p[i].y; float sz0 = p[i].z;

    int j = blockIdx.x / BLOCK_NUM;
    sp[3 * threadIdx.x + 0] = p[j * BLOCK_SIZE + threadIdx.x].x;
    sp[3 * threadIdx.x + 1] = p[j * BLOCK_SIZE + threadIdx.x].y;
    sp[3 * threadIdx.x + 2] = p[j * BLOCK_SIZE + threadIdx.x].z;
    __syncthreads();
#pragma unroll 32
    for(int k = 0; k < BLOCK_SIZE; k++){
	float dx = sp[3 * k + 0] - sx0;
	float dy = sp[3 * k + 1] - sy0;
	float dz = sp[3 * k + 2] - sz0;
	float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
	float invDist = rsqrtf(distSqr);
	float invDist3 = invDist * invDist * invDist;
	Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }
    atomicAdd(&p[i].vx, dt*Fx);
    atomicAdd(&p[i].vy, dt*Fy);
    atomicAdd(&p[i].vz, dt*Fz);
    atomicSub(&state[i], 1);
    if(!state[i]){
	p[i].x = sx0 + p[i].vx * dt;
	p[i].y = sy0 + p[i].vy * dt;
	p[i].z = sz0 + p[i].vz * dt;
	atomicExch(&state[i], BLOCK_TILE);
    }
}


int main(const int argc, const char** argv) {

    /*
     * Do not change the value for `nBodies` here. If you would like to modify it,
     * pass values into the command line.
     */

    int nBodies = 2<<11;
    int salt = 0;
    if (argc > 1) nBodies = 2<<atoi(argv[1]);

    /*
     * This salt is for assessment reasons. Tampering with it will result in automatic failure.
     */

    if (argc > 2) salt = atoi(argv[2]);

    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;
    int *h_state;
    int *d_state;

    buf = (float *)malloc(bytes);
    h_state = (int *)malloc(nBodies*sizeof(int));
    for(int i = 0; i < nBodies; i++)
        h_state[i] = BLOCK_TILE;

    Body *h_p = (Body*)buf;
    Body *d_p;

    size_t threadsPerBlock = BLOCK_SIZE;
    size_t numberOfBlocks = BLOCK_NUM;



    /*
     * As a constraint of this exercise, `randomizeBodies` must remain a host function.
     */

    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

    hipMalloc((void**)&d_p, bytes);
    hipMalloc((void**)&d_state, nBodies*sizeof(int));
    hipMemcpy(d_p, h_p, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_state, h_state, nBodies*sizeof(int), hipMemcpyHostToDevice);

    double totalTime = 0.0;

    /*
     * This simulation will run for 10 cycles of time, calculating gravitational
     * interaction amongst bodies, and adjusting their positions to reflect.
     */

    /*******************************************************************/
    // Do not modify these 2 lines of code.


    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();
        /*******************************************************************/

        /*
         * You will likely wish to refactor the work being done in `bodyForce`,
         * as well as the work to integrate the positions.
         */

        bodyForce<<<numberOfBlocks*BLOCK_TILE, threadsPerBlock, 3*BLOCK_SIZE*sizeof(float)>>>(d_p, d_state, dt, nBodies); // compute interbody forces

        /*
         * This position integration cannot occur until this round of `bodyForce` has completed.
         * Also, the next round of `bodyForce` cannot begin until the integration is complete.
         */

        if(iter == nIters - 1)hipMemcpy(h_p, d_p, bytes, hipMemcpyDeviceToHost);

        /*******************************************************************/
        // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/

    /*
     * Feel free to modify code below.
     */

    free(buf);
    hipFree(d_p);
}

